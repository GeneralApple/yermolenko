#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <ctime>
#include <cstdlib>

using namespace std;

__global__ void Random(int *c,int r)
{
  
  
  unsigned int ind = blockIdx.x*blockDim.x+threadIdx.x;
  unsigned long int tmp = rnd0 * (1+1664524*ind) + 1013904223UL*ind;
  
  c[ind] = 1 + tmp%100;
}


int main()
{
int N = 1000;
  int *y, *d_y;
  y = (int*)malloc(N*sizeof(int));
  hipMalloc(&d_y, N*sizeof(int));
  
  srand(time(0));
	int rnd0 = 1 + rand() % 100;
  
  int *d_rnd0;
hipMalloc(&d_rnd0, N*sizeof(int));
  
  hipMemcpy(d_rnd0, &rnd0, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(int), hipMemcpyHostToDevice);
  
   Random<<<(N+255)/256, 256>>>(d_y, *d_rnd0);
	
  hipMemcpy(y, d_y, N*sizeof(int), hipMemcpyDeviceToHost);	
	
	for (int i = 0; i < N; i++)
  {
   
   cout<<y[i]<<endl;
  }
	
for(int j = 0;j<10;j++)
{ int n = 0;
 	for(int k = 0; k<N;k++){
		if(y[k]>10*j && y[k]<=10*(j+1)) n++ }
 cout<<n<<endl;
}
	
hipFree(d_y);

	
 free(y);
