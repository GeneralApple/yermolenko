#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <ctime>
#include <cstdlib>

using namespace std;

__global__ 
void Random(int *c,int r)
{
  
  
  unsigned int ind = blockIdx.x*blockDim.x+threadIdx.x;
  unsigned long int tmp = rnd0 * (1+1664524*ind) + 1013904223UL*ind;
  /*hiprandState_t state;
  hiprand_init(ind, /* the seed controls the sequence of random values that are produced */
              0, /* the sequence number is only important with multiple cores */
              0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
             /* &state);*/
  c[ind] = 1 + tmp%100;
}


int main()
{
int N = 1000;
  int *y, *d_y;
  y = (int*)malloc(N*sizeof(int));
  hipMalloc(&d_y, N*sizeof(int));
  
  srand(time(0));
	int rnd0 = 1 + rand() % 100;
  
  int *d_rnd0;
  
  hipMemcpy(d_rnd0, &rnd0, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(int), hipMemcpyHostToDevice);
  
