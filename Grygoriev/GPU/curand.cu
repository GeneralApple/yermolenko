#include "hip/hip_runtime.h"
#include <unistd.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>


__global__ void init(unsigned int seed, hiprandState_t* states) {

  /* we have to initialize the state */
  hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
              blockIdx.x*blockDim.x+threadIdx.x, /* the sequence number should be different for each core (unless you want all
                             cores to get the same sequence of numbers for some reason - use thread id! */
              0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
              &states[blockIdx.x*blockDim.x+threadIdx.x]);
}
__global__ void Random(hiprandState_t* states, unsigned int *c)
{
  unsigned int ind = blockIdx.x*blockDim.x+threadIdx.x;
 
  c[ind] = 1 + hiprand(&states)%100;
}

int main(void)
{
  
   hiprandState_t* states;
  hipMalloc((void**) &states, N * sizeof(hiprandState_t));
  
  int N = 1000000;
  
   init<<<N/1000, 1000>>>(time(0), states);
  
  
  int *y, *d_y;
  
  y = (int*)malloc(N*sizeof(int));

  hipMalloc((void**) &d_y, N*sizeof(int));
  


  for (int i = 0; i < N; i++) {
    y[i] = 0;
  }
  //hipMemcpy(d_y, y, N*sizeof(int), hipMemcpyHostToDevice);

   Random<<<(1000, 1000>>>(states,d_y);

  hipMemcpy(y, d_y, N*sizeof(int), hipMemcpyDeviceToHost);

 
 
	for (int i = 0; i < N; i++)
  {
   
   cout<<y[i]<<endl;
  }
	
/*for(int j = 0;j<1000;j++)
{ int n = 0;
 	for(int k = 0; k<N;k++){
		if(y[k]>1000*j && y[k]<=1000*(j+1)) n++ }
 cout<<n<<endl;
}*/
  

  hipFree(d_y);
  free(y);
             
   return 0;
} 
