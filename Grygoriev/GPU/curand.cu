#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


__global__ void init(unsigned int seed, hiprandState_t* states) {

  /* we have to initialize the state */
  hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
              blockIdx.x*blockDim.x+threadIdx.x, /* the sequence number should be different for each core (unless you want all
                             cores to get the same sequence of numbers for some reason - use thread id! */
              0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
              &states[blockIdx.x]);
}
__global__ void Random(hiprandState_t* states, unsigned int *c)
{
  unsigned int ind = blockIdx.x*blockDim.x+threadIdx.x;
 
  c[ind] = 1 + hiprand(&states)%100;
}

int main(void)
{
  
   hiprandState_t* states;
  hipMalloc((void**) &states, N * sizeof(hiprandState_t));
   init<<<(N+255)/256, 256>>>(time(0), states);
  int N = 1000000;
  int *y, *d_y;
  
  y = (int*)malloc(N*sizeof(int));

  hipMalloc(&d_y, N*sizeof(int));
  


  for (int i = 0; i < N; i++) {
    y[i] = 0;
  }
  //hipMemcpy(d_y, y, N*sizeof(int), hipMemcpyHostToDevice);

   Random<<<(N+255)/256, 256>>>(d_y);

  hipMemcpy(y, d_y, N*sizeof(int), hipMemcpyDeviceToHost);

  //int maxError = 0;
  for (int i = 0; i < N; i++)
  {
   // maxError = max(maxError, abs(y[i]-4));
   printf("Rand is: %d\n", y[i]);
  }
  //printf("Max error: %d  %d\n", maxError, N);

  hipFree(d_y);
  free(y);
} 
