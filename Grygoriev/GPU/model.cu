#include "hip/hip_runtime.h"
#include ""

#include <iostrem>
#include <time.h>
#include <cmath>
#include <cstdlib>

#define pi    3.1416
using namespace std;


int main() {

int N;
cin >> N;
 
double mu=0.07;

double h = 100;
  
  int *dN;
  
  double *dmu, *dh;
  
  hipMalloc((void**)&dN, sizeof(int));
  hipMalloc((void**)&dmu, sizeof(double));
  hipMalloc((void**)&dh, sizeof(double));
  
   hipMemcpy(dN, &N, sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(dmu, &mu, sizeof(double), hipMemcpyHostToDevice);
   hipMemcpy(dh, &h, sizeof(double), hipMemcpyHostToDevice);
  


  

//double position[N];


