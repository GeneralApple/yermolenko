#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostrem>
#include <time.h>
#include <cmath>
#include <cstdlib>

//#define pi    3.1416
using namespace std;

__global__ void setup_kernel ( hiprandState * state, unsigned long seed )
{
    unsigned int id = blockIdx.x*blockDim.x+threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );
}

__device__ float expo(double mu, hiprandState* globalState, int ind)
{
  do{ double u = generate(globalState, ind);}
    while(u!=0);
    
  return -log(1- u) / mu;}

__device__ float generate( hiprandState* globalState, int ind ) 
{
    //int ind = threadIdx.x;
    hiprandState localState = globalState[ind];
    float RANDOM = hiprand_uniform( &localState );
    globalState[ind] = localState;
    return RANDOM;
}

__global__ void model(double *pos, double *dmu, double *dh, hiprandState* globalState)
{
    
     unsigned int id = blockIdx.x*blockDim.x+threadIdx.x;
    
    double mu = *dmu;
   double  h = *dh;
    
    


double x=0, y=0; //початкові координати частинки
    
    double l; //відстань прольоту частинки між актами взаємодії

//int n = 0;

double Theta = 0; //напрям руху частинки
    
    double dTh;
    
while (y<d)
{
  l = expo(mu, globalState, id);
    
    
 y = y + l*cos(Theta);
 x = x + l*sin(Theta);
    
    if(y<0)
{x = h*10;
break;}
    
    
    if(x<-h) break;

if(y>h) break;
    
    
    dTh = generate(globalState, id)*(4*pi)-(2*pi);

    Theta = Theta + dTh;
}
    
    pos[id] = x;
}
    
int main() {

int N;
cin >> N;
    
  double d;
    cin >> d;
 
double mu=0.07;

double h = 100;
  
  int *dN;
  
  double *dmu, *dh, *dd;
  
  //hipMalloc((void**)&dN, sizeof(int));
  hipMalloc((void**)&dmu, sizeof(double));
  hipMalloc((void**)&dh, sizeof(double));
    hipMalloc((void**)&dd, sizeof(double));
  
   //hipMemcpy(dN, &N, sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(dmu, &mu, sizeof(double), hipMemcpyHostToDevice);
   hipMemcpy(dh, &h, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dd, &d, sizeof(double), hipMemcpyHostToDevice);
  


  double xpos[N];
  double *pos;
 
 hipMalloc((void**) &pos, N*sizeof(int));
 
 hiprandState* devStates;
 hipMalloc ( &devStates, N*sizeof( hiprandState ) );
 
setup_kernel <<< N/1000, 1000 >>> ( devStates,unsigned(time(NULL)) );
 


