#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <time.h>
#include <cmath>
#include <cstdlib>
#include<hiprand.h>
#include<hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <stdio.h>


using namespace std;

__global__ void setup_kernel ( hiprandState * state, unsigned long seed )
{
unsigned int id = blockIdx.x*blockDim.x+threadIdx.x;
hiprand_init ( seed, id, 0, &state[id] );
}



__device__ float generate( hiprandState* globalState, int ind )
{
    //int ind = threadIdx.x;
    hiprandState localState = globalState[ind];
    float RANDOM = hiprand_uniform( &localState );
    globalState[ind] = localState;
    return RANDOM;
}


__device__ float expo(double mu, hiprandState* globalState, int ind)
{
    double u;
    do{
        u = generate(globalState, ind);
    } while(u==1.0);
    
    return -log(1- u) * mu;
}


__global__ void model(double *dd, double *pos, double *dmu, double *dh, hiprandState* globalState)
{
unsigned int id = blockIdx.x*blockDim.x+threadIdx.x;

double mu = *dmu;
double h = *dh;
double d = *dd;
double pi = 3.1416;
double x=0, y=0; //початкові координати частинки
double l; //відстань прольоту частинки між актами взаємодії
//int n = 0;
double Theta = 0; //напрям руху частинки
double dTh;
int n = 0;
while (n<10)
{
    l = expo(mu, globalState, id);
    
    y = y + l*cos(Theta);
    x = x + l*sin(Theta);
        if(y>=d) break;
    if(y<0)
    {x = h*10;
    break;}
    if(x<-h) break;
    if(y>h) break;
    dTh = generate(globalState, id)*(4*pi)-(2*pi);
    Theta = Theta + dTh;
n++;

}

//printf("position: %f \n", x);
pos[id] = x;
}



int main() {
int N = 1; //кількість частинок
//cin >> N;
double d; //товщина пластинки
cin >> d;
double mu=0.07; //довжина вільного пробігу
double h = 100;
double *dmu, *dh, *dd;

//hipMalloc((void**)&dN, sizeof(int));
hipMalloc((void**)&dmu, sizeof(double));
hipMalloc((void**)&dh, sizeof(double));
hipMalloc((void**)&dd, sizeof(double));
//hipMemcpy(dN, &N, sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(dmu, &mu, sizeof(double), hipMemcpyHostToDevice);
hipMemcpy(dh, &h, sizeof(double), hipMemcpyHostToDevice);
hipMemcpy(dd, &d, sizeof(double), hipMemcpyHostToDevice);
double xpos[N];
double *pos;
hipMalloc((void**) &pos, N*sizeof(double));
hiprandState* devStates;
hipMalloc ( &devStates, N*sizeof( hiprandState ) );

setup_kernel <<< 1, 1 >>> ( devStates,unsigned(time(NULL)) );

model <<< 10, 1 >>> (dd,pos,dmu,dh, devStates);

hipMemcpy(xpos, pos, N*sizeof(double), hipMemcpyDeviceToHost);

for (int j = 0; j<20; j++)
{
    int n = 0;
    for(int k = 0; k<N;k++){
    
    if( (0.1*j -1)<=xpos[k] && (0.1*j -0.9) > xpos[k]) n++;
     }
cout<<n<<endl;
}

    //запис в файл
	ofstream f;

	f.open("E:\\data.txt");
	for (int i = 0; i < N; i++) f << xpos[i] << endl;
	f.close();
    
return 0;}
