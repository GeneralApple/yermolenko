#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostrem>
#include <time.h>
#include <cmath>
#include <cstdlib>

//#define pi    3.1416
using namespace std;

__global__ void setup_kernel ( hiprandState * state, unsigned long seed )
{
    unsigned int id = blockIdx.x*blockDim.x+threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );
}



int main() {

int N;
cin >> N;
 
double mu=0.07;

double h = 100;
  
  int *dN;
  
  double *dmu, *dh;
  
  hipMalloc((void**)&dN, sizeof(int));
  hipMalloc((void**)&dmu, sizeof(double));
  hipMalloc((void**)&dh, sizeof(double));
  
   hipMemcpy(dN, &N, sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(dmu, &mu, sizeof(double), hipMemcpyHostToDevice);
   hipMemcpy(dh, &h, sizeof(double), hipMemcpyHostToDevice);
  


  double pos[N];
  double *dpos;
 
 hipMalloc((void**) &dpos, N*sizeof(int));
 
 hiprandState* devStates;
 hipMalloc ( &devStates, N*sizeof( hiprandState ) );
 
setup_kernel <<< N/1000, 1000 >>> ( devStates,unsigned(time(NULL)) );
 


